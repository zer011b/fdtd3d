#define CUDA_SOURCES

#include "InternalSchemeHelper.h"

#ifdef CUDA_ENABLED

template <SchemeType_t Type, template <typename, bool> class TCoord, LayoutType layout_type, template <typename> class TGrid>
CUDA_HOST
void
InternalSchemeHelper::allocateGridsOnGPU (InternalSchemeBase<Type, TCoord, layout_type, TGrid> *gpuScheme)
{
  typedef TCoord<grid_coord, true> TC;
  typedef TCoord<grid_coord, false> TCS;
  typedef TCoord<FPValue, true> TCFP;
  typedef TCoord<FPValue, false> TCSFP;

  cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Eps, sizeof(TGrid<TC>)));
  cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Mu, sizeof(TGrid<TC>)));

  if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Ex, sizeof(TGrid<TC>))); }
  if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Ey, sizeof(TGrid<TC>))); }
  if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Ez, sizeof(TGrid<TC>))); }
  if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Hx, sizeof(TGrid<TC>))); }
  if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Hy, sizeof(TGrid<TC>))); }
  if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Hz, sizeof(TGrid<TC>))); }

  if (SOLVER_SETTINGS.getDoUsePML ())
  {
    if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Dx, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Dy, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Dz, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Bx, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->By, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->Bz, sizeof(TGrid<TC>))); }

    if (SOLVER_SETTINGS.getDoUseMetamaterials ())
    {
      if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->D1x, sizeof(TGrid<TC>))); }
      if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->D1y, sizeof(TGrid<TC>))); }
      if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->D1z, sizeof(TGrid<TC>))); }
      if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->B1x, sizeof(TGrid<TC>))); }
      if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->B1y, sizeof(TGrid<TC>))); }
      if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->B1z, sizeof(TGrid<TC>))); }
    }

    if (gpuScheme->doNeedSigmaX) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->SigmaX, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedSigmaY) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->SigmaY, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedSigmaZ) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->SigmaZ, sizeof(TGrid<TC>))); }
  }

  if (SOLVER_SETTINGS.getDoUseAmplitudeMode ())
  {
    if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->ExAmplitude, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->EyAmplitude, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->EzAmplitude, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->HxAmplitude, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->HyAmplitude, sizeof(TGrid<TC>))); }
    if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->HzAmplitude, sizeof(TGrid<TC>))); }
  }

  if (SOLVER_SETTINGS.getDoUseMetamaterials ())
  {
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->OmegaPE, sizeof(TGrid<TC>)));
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->GammaE, sizeof(TGrid<TC>)));
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->OmegaPM, sizeof(TGrid<TC>)));
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->GammaM, sizeof(TGrid<TC>)));
  }

  if (SOLVER_SETTINGS.getDoUseTFSF ())
  {
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->EInc, sizeof(TGrid<GridCoordinate1D>)));
    cudaCheckErrorCmd (hipMalloc ((void **) &gpuScheme->HInc, sizeof(TGrid<GridCoordinate1D>)));
  }
}

template <SchemeType_t Type, template <typename, bool> class TCoord, LayoutType layout_type, template <typename> class TGrid>
CUDA_HOST
void
InternalSchemeHelper::copyGridsToGPU (InternalSchemeBase<Type, TCoord, layout_type, TGrid> *intScheme,
                                      InternalSchemeBase<Type, TCoord, layout_type, TGrid> *gpuScheme)
{
  typedef TCoord<grid_coord, true> TC;
  typedef TCoord<grid_coord, false> TCS;
  typedef TCoord<FPValue, true> TCFP;
  typedef TCoord<FPValue, false> TCSFP;

  cudaCheckErrorCmd (hipMemcpy (gpuScheme->Eps, intScheme->Eps, sizeof(TGrid<TC>), hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (gpuScheme->Mu, intScheme->Mu, sizeof(TGrid<TC>), hipMemcpyHostToDevice));

  if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Ex, intScheme->Ex, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Ey, intScheme->Ey, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Ez, intScheme->Ez, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Hx, intScheme->Hx, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Hy, intScheme->Hy, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Hz, intScheme->Hz, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }

  if (SOLVER_SETTINGS.getDoUsePML ())
  {
    if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Dx, intScheme->Dx, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Dy, intScheme->Dy, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Dz, intScheme->Dz, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Bx, intScheme->Bx, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->By, intScheme->By, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->Bz, intScheme->Bz, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }

    if (SOLVER_SETTINGS.getDoUseMetamaterials ())
    {
      if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->D1x, intScheme->D1x, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
      if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->D1y, intScheme->D1y, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
      if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->D1z, intScheme->D1z, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
      if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->B1x, intScheme->B1x, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
      if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->B1y, intScheme->B1y, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
      if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->B1z, intScheme->B1z, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    }

    if (gpuScheme->doNeedSigmaX) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->SigmaX, intScheme->SigmaX, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedSigmaY) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->SigmaY, intScheme->SigmaY, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedSigmaZ) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->SigmaZ, intScheme->SigmaZ, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  }

  if (SOLVER_SETTINGS.getDoUseAmplitudeMode ())
  {
    if (gpuScheme->doNeedEx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->ExAmplitude, intScheme->ExAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedEy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->EyAmplitude, intScheme->EyAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedEz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->EzAmplitude, intScheme->EzAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHx) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->HxAmplitude, intScheme->HxAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHy) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->HyAmplitude, intScheme->HyAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
    if (gpuScheme->doNeedHz) { cudaCheckErrorCmd (hipMemcpy (gpuScheme->HzAmplitude, intScheme->HzAmplitude, sizeof(TGrid<TC>), hipMemcpyHostToDevice)); }
  }

  if (SOLVER_SETTINGS.getDoUseMetamaterials ())
  {
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->OmegaPE, intScheme->OmegaPE, sizeof(TGrid<TC>), hipMemcpyHostToDevice));
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->GammaE, intScheme->GammaE, sizeof(TGrid<TC>), hipMemcpyHostToDevice));
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->OmegaPM, intScheme->OmegaPM, sizeof(TGrid<TC>), hipMemcpyHostToDevice));
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->GammaM, intScheme->GammaM, sizeof(TGrid<TC>), hipMemcpyHostToDevice));
  }

  if (SOLVER_SETTINGS.getDoUseTFSF ())
  {
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->EInc, intScheme->EInc, sizeof(TGrid<GridCoordinate1D>), hipMemcpyHostToDevice));
    cudaCheckErrorCmd (hipMemcpy (gpuScheme->HInc, intScheme->HInc, sizeof(TGrid<GridCoordinate1D>), hipMemcpyHostToDevice));
  }
}

#endif /* CUDA_ENABLED */
