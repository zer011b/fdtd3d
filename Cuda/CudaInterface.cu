#include "CudaInterface.h"
#include "CudaGlobalKernels.h"

void cudaExecute2DTMzSteps (CudaExitStatus *retval,
                            FieldValue *Ez, FieldValue *Hx, FieldValue *Hy,
                            FieldValue *Ez_prev, FieldValue *Hx_prev, FieldValue *Hy_prev,
                            FieldValue *eps, FieldValue *mu,
                            FieldValue gridTimeStep, FieldValue gridStep,
                            grid_coord sx_Ez, grid_coord sy_Ez,
                            grid_coord sx_Hx, grid_coord sy_Hx,
                            grid_coord sx_Hy, grid_coord sy_Hy,
                            grid_iter sizeEps, grid_iter sizeMu,
                            time_step stepStart, time_step stepEnd,
                            uint32_t blocksX, uint32_t blocksY, uint32_t threadsX, uint32_t threadsY)
{
  FieldValue *Ez_cuda;
  FieldValue *Hx_cuda;
  FieldValue *Hy_cuda;

  FieldValue *Ez_cuda_prev;
  FieldValue *Hx_cuda_prev;
  FieldValue *Hy_cuda_prev;

  FieldValue *eps_cuda;
  FieldValue *mu_cuda;

  grid_iter sizeEz = (grid_iter) sx_Ez * sy_Ez * sizeof (FieldValue);
  grid_iter sizeHx = (grid_iter) sx_Hx * sy_Hx * sizeof (FieldValue);
  grid_iter sizeHy = (grid_iter) sx_Hy * sy_Hy * sizeof (FieldValue);
  //printf ("%llu=%ld*%ld*%lld", size, sx, sy, sizeof (FieldValue));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda, sizeEz));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda, sizeHx));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda, sizeHy));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda_prev, sizeEz));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda_prev, sizeHx));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda_prev, sizeHy));

  cudaCheckErrorCmd (hipMalloc ((void **) &eps_cuda, sizeEps));
  cudaCheckErrorCmd (hipMalloc ((void **) &mu_cuda, sizeMu));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda, Ez, sizeEz, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda, Hx, sizeHx, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda, Hy, sizeHy, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda_prev, Ez_prev, sizeEz, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda_prev, Hx_prev, sizeHx, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda_prev, Hy_prev, sizeHy, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (eps_cuda, eps, sizeEps, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (mu_cuda, mu, sizeMu, hipMemcpyHostToDevice));

  dim3 blocks (blocksX, blocksY);
  dim3 threads (threadsX, threadsY);

  CudaExitStatus exitStatus;
  CudaExitStatus *exitStatusCuda;
  cudaCheckErrorCmd (hipMalloc ((void **) &exitStatusCuda, sizeof (CudaExitStatus)));

  for (time_step t = stepStart; t < stepEnd; ++t)
  {
    cudaCheckExitStatus (cudaCalculateTMzEStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Ez_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        eps_cuda,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx_Ez,
                                                                        sy_Ez,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzESource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Ez_cuda_prev,
                                                                          sx_Ez,
                                                                          sy_Ez,
                                                                          t));

    cudaCheckExitStatus (cudaCalculateTMzHStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Hx_cuda,
                                                                        Hy_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        mu_cuda,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx_Hx,
                                                                        sy_Hx,
                                                                        sx_Hy,
                                                                        sy_Hy,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzHSource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Hx_cuda_prev,
                                                                          Hy_cuda_prev,
                                                                          sx_Hx,
                                                                          sy_Hx,
                                                                          sx_Hy,
                                                                          sy_Hy,
                                                                          t));
  }

  cudaCheckErrorCmd (hipMemcpy (Ez, Ez_cuda, sizeEz, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx, Hx_cuda, sizeHx, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy, Hy_cuda, sizeHy, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (Ez_prev, Ez_cuda_prev, sizeEz, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx_prev, Hx_cuda_prev, sizeHx, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy_prev, Hy_cuda_prev, sizeHy, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (eps, eps_cuda, sizeEps, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (mu, mu_cuda, sizeMu, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipFree (Ez_cuda));
  cudaCheckErrorCmd (hipFree (Hx_cuda));
  cudaCheckErrorCmd (hipFree (Hy_cuda));

  cudaCheckErrorCmd (hipFree (Ez_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hx_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hy_cuda_prev));

  cudaCheckErrorCmd (hipFree (eps_cuda));
  cudaCheckErrorCmd (hipFree (mu_cuda));

  *retval = CUDA_OK;
  return;
}
