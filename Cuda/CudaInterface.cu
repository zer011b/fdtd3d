#include "CudaInterface.h"
#include "CudaGlobalKernels.h"

#ifdef PARALLEL_GRID
void cudaExecute2DTMzSteps (CudaExitStatus *retval,
                            YeeGridLayout &yeeLayout,
                            FieldValue gridTimeStep, FieldValue gridStep,
                            ParallelGrid &Ez,
                            ParallelGrid &Hx,
                            ParallelGrid &Hy,
                            ParallelGrid &Eps,
                            ParallelGrid &Mu,
                            time_step stepStart, time_step stepEnd)
{
  ParallelGridCoordinate EzSizeCoord = Ez.getSize ();
  ParallelGridCoordinate HxSizeCoord = Hx.getSize ();
  ParallelGridCoordinate HySizeCoord = Hy.getSize ();
  ParallelGridCoordinate EpsSizeCoord = Eps.getSize ();
  ParallelGridCoordinate MuSizeCoord = Mu.getSize ();

  grid_iter sizeEz = EzSizeCoord.calculateTotalCoord();
  grid_iter sizeHx = HxSizeCoord.calculateTotalCoord();
  grid_iter sizeHy = HySizeCoord.calculateTotalCoord();
  grid_iter sizeEps = EpsSizeCoord.calculateTotalCoord();
  grid_iter sizeMu = MuSizeCoord.calculateTotalCoord();

  grid_iter sizeEzRaw = (grid_iter) sizeEz * sizeof (FieldValue);
  grid_iter sizeHxRaw = (grid_iter) sizeHx * sizeof (FieldValue);
  grid_iter sizeHyRaw = (grid_iter) sizeHy * sizeof (FieldValue);
  grid_iter sizeEpsRaw = (grid_iter) sizeEps * sizeof (FieldValue);
  grid_iter sizeMuRaw = (grid_iter) sizeMu * sizeof (FieldValue);

  FieldValue *tmp_Ez = new FieldValue [sizeEz];
  FieldValue *tmp_Hx = new FieldValue [sizeHx];
  FieldValue *tmp_Hy = new FieldValue [sizeHy];

  FieldValue *tmp_Ez_prev = new FieldValue [sizeEz];
  FieldValue *tmp_Hx_prev = new FieldValue [sizeHx];
  FieldValue *tmp_Hy_prev = new FieldValue [sizeHy];

  FieldValue *tmp_eps = new FieldValue [sizeEps];
  FieldValue *tmp_mu = new FieldValue [sizeMu];

  for (grid_iter i = 0; i < sizeEz; ++i)
  {
    FieldPointValue* valEz = Ez.getFieldPointValue (i);
    tmp_Ez[i] = valEz->getCurValue ();
    tmp_Ez_prev[i] = valEz->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeHx; ++i)
  {
    FieldPointValue* valHx = Hx.getFieldPointValue (i);
    tmp_Hx[i] = valHx->getCurValue ();
    tmp_Hx_prev[i] = valHx->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeHy; ++i)
  {
    FieldPointValue* valHy = Hy.getFieldPointValue (i);
    tmp_Hy[i] = valHy->getCurValue ();
    tmp_Hy_prev[i] = valHy->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeEps; ++i)
  {
    FieldPointValue *valEps = Eps.getFieldPointValue (i);
    tmp_eps[i] = valEps->getCurValue ();
  }

  for (grid_iter i = 0; i < sizeMu; ++i)
  {
    FieldPointValue *valMu = Mu.getFieldPointValue (i);
    tmp_mu[i] = valMu->getCurValue ();
  }

  FieldValue *Ez_cuda;
  FieldValue *Hx_cuda;
  FieldValue *Hy_cuda;

  FieldValue *Ez_cuda_prev;
  FieldValue *Hx_cuda_prev;
  FieldValue *Hy_cuda_prev;

  FieldValue *eps_cuda;
  FieldValue *mu_cuda;

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda, sizeEzRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda, sizeHxRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda, sizeHyRaw));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda_prev, sizeEzRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda_prev, sizeHxRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda_prev, sizeHyRaw));

  cudaCheckErrorCmd (hipMalloc ((void **) &eps_cuda, sizeEpsRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &mu_cuda, sizeMuRaw));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda, tmp_Ez, sizeEzRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda, tmp_Hx, sizeHxRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda, tmp_Hy, sizeHyRaw, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda_prev, tmp_Ez_prev, sizeEzRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda_prev, tmp_Hx_prev, sizeHxRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda_prev, tmp_Hy_prev, sizeHyRaw, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (eps_cuda, tmp_eps, sizeEpsRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (mu_cuda, tmp_mu, sizeMuRaw, hipMemcpyHostToDevice));

  CudaExitStatus exitStatus;
  CudaExitStatus *exitStatusCuda;
  cudaCheckErrorCmd (hipMalloc ((void **) &exitStatusCuda, sizeof (CudaExitStatus)));

  dim3 blocksEz (EzSizeCoord.getX () / 16, EzSizeCoord.getY () / 16);
  dim3 threadsEz (16, 16);

  dim3 blocksHx (HxSizeCoord.getX () / 16, HxSizeCoord.getY () / 16);
  dim3 threadsHx (16, 16);

  dim3 blocksHy (HySizeCoord.getX () / 16, HySizeCoord.getY () / 16);
  dim3 threadsHy (16, 16);

  for (time_step t = stepStart; t < stepEnd; ++t)
  {
    GridCoordinate3D EzStart = yeeLayout.getEzStart (Ez.getStart ());
    GridCoordinate3D EzEnd = yeeLayout.getEzEnd (Ez.getEnd ());

    GridCoordinate3D HxStart = yeeLayout.getHxStart (Hx.getStart ());
    GridCoordinate3D HxEnd = yeeLayout.getHxEnd (Hx.getEnd ());

    GridCoordinate3D HyStart = yeeLayout.getHyStart (Hy.getStart ());
    GridCoordinate3D HyEnd = yeeLayout.getHyEnd (Hy.getEnd ());

    cudaCheckExitStatus (cudaCalculateTMzEzStep <<< blocksEz, threadsEz >>> (exitStatusCuda,
                                                                             Ez_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hx_cuda_prev,
                                                                             Hy_cuda_prev,
                                                                             eps_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             EzStart.getX (),
                                                                             EzStart.getY (),
                                                                             EzEnd.getX (),
                                                                             EzEnd.getY (),
                                                                             EzSizeCoord.getX (),
                                                                             EzSizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzEzSource <<< blocksEz, threadsEz >>> (exitStatusCuda,
                                                                               Ez_cuda_prev,
                                                                               EzStart.getX (),
                                                                               EzStart.getY (),
                                                                               EzEnd.getX (),
                                                                               EzEnd.getY (),
                                                                               EzSizeCoord.getX (),
                                                                               EzSizeCoord.getY (),
                                                                               t));

    Ez.nextShareStep ();

    cudaCheckExitStatus (cudaCalculateTMzHxStep <<< blocksHx, threadsHx >>> (exitStatusCuda,
                                                                             Hx_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hx_cuda_prev,
                                                                             mu_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             HxStart.getX (),
                                                                             HxStart.getY (),
                                                                             HxEnd.getX (),
                                                                             HxEnd.getY (),
                                                                             HxSizeCoord.getX (),
                                                                             HxSizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzHyStep <<< blocksHy, threadsHy >>> (exitStatusCuda,
                                                                             Hy_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hy_cuda_prev,
                                                                             mu_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             HyStart.getX (),
                                                                             HyStart.getY (),
                                                                             HyEnd.getX (),
                                                                             HyEnd.getY (),
                                                                             HySizeCoord.getX (),
                                                                             HySizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzHxSource <<< blocksHx, threadsHx >>> (exitStatusCuda,
                                                                               Hx_cuda_prev,
                                                                               HxStart.getX (),
                                                                               HxStart.getY (),
                                                                               HxEnd.getX (),
                                                                               HxEnd.getY (),
                                                                               HxSizeCoord.getX (),
                                                                               HxSizeCoord.getY (),
                                                                               t));

    cudaCheckExitStatus (cudaCalculateTMzHySource <<< blocksHy, threadsHy >>> (exitStatusCuda,
                                                                               Hy_cuda_prev,
                                                                               HyStart.getX (),
                                                                               HyStart.getY (),
                                                                               HyEnd.getX (),
                                                                               HyEnd.getY (),
                                                                               HySizeCoord.getX (),
                                                                               HySizeCoord.getY (),
                                                                               t));

    Hx.nextShareStep ();
    Hy.nextShareStep ();
  }

  cudaCheckErrorCmd (hipMemcpy (tmp_Ez, Ez_cuda, sizeEzRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hx, Hx_cuda, sizeHxRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hy, Hy_cuda, sizeHyRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (tmp_Ez_prev, Ez_cuda_prev, sizeEzRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hx_prev, Hx_cuda_prev, sizeHxRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hy_prev, Hy_cuda_prev, sizeHyRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (tmp_eps, eps_cuda, sizeEpsRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_mu, mu_cuda, sizeMuRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipFree (Ez_cuda));
  cudaCheckErrorCmd (hipFree (Hx_cuda));
  cudaCheckErrorCmd (hipFree (Hy_cuda));

  cudaCheckErrorCmd (hipFree (Ez_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hx_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hy_cuda_prev));

  cudaCheckErrorCmd (hipFree (eps_cuda));
  cudaCheckErrorCmd (hipFree (mu_cuda));

  for (grid_iter i = 0; i < sizeEz; ++i)
  {
    FieldPointValue* valEz = Ez.getFieldPointValue (i);
    valEz->setCurValue (tmp_Ez[i]);
    valEz->setPrevValue (tmp_Ez_prev[i]);
  }

  for (grid_iter i = 0; i < sizeHx; ++i)
  {
    FieldPointValue* valHx = Hx.getFieldPointValue (i);
    valHx->setCurValue (tmp_Hx[i]);
    valHx->setPrevValue (tmp_Hx_prev[i]);
  }

  for (grid_iter i = 0; i < sizeHy; ++i)
  {
    FieldPointValue* valHy = Hy.getFieldPointValue (i);
    valHy->setCurValue (tmp_Hy[i]);
    valHy->setPrevValue (tmp_Hy_prev[i]);
  }

  delete[] tmp_Ez;
  delete[] tmp_Hx;
  delete[] tmp_Hy;

  delete[] tmp_Ez_prev;
  delete[] tmp_Hx_prev;
  delete[] tmp_Hy_prev;

  delete[] tmp_eps;
  delete[] tmp_mu;

  *retval = CUDA_OK;
  return;
}

#else

void cudaExecute2DTMzSteps (CudaExitStatus *retval,
                            YeeGridLayout &yeeLayout,
                            FieldValue gridTimeStep, FieldValue gridStep,
                            Grid<GridCoordinate2D> &Ez,
                            Grid<GridCoordinate2D> &Hx,
                            Grid<GridCoordinate2D> &Hy,
                            Grid<GridCoordinate2D> &Eps,
                            Grid<GridCoordinate2D> &Mu,
                            time_step stepStart, time_step stepEnd)
{
  GridCoordinate2D EzSizeCoord = Ez.getSize ();
  GridCoordinate2D HxSizeCoord = Hx.getSize ();
  GridCoordinate2D HySizeCoord = Hy.getSize ();
  GridCoordinate2D EpsSizeCoord = Eps.getSize ();
  GridCoordinate2D MuSizeCoord = Mu.getSize ();

  grid_iter sizeEz = EzSizeCoord.calculateTotalCoord();
  grid_iter sizeHx = HxSizeCoord.calculateTotalCoord();
  grid_iter sizeHy = HySizeCoord.calculateTotalCoord();
  grid_iter sizeEps = EpsSizeCoord.calculateTotalCoord();
  grid_iter sizeMu = MuSizeCoord.calculateTotalCoord();

  grid_iter sizeEzRaw = (grid_iter) sizeEz * sizeof (FieldValue);
  grid_iter sizeHxRaw = (grid_iter) sizeHx * sizeof (FieldValue);
  grid_iter sizeHyRaw = (grid_iter) sizeHy * sizeof (FieldValue);
  grid_iter sizeEpsRaw = (grid_iter) sizeEps * sizeof (FieldValue);
  grid_iter sizeMuRaw = (grid_iter) sizeMu * sizeof (FieldValue);

  FieldValue *tmp_Ez = new FieldValue [sizeEz];
  FieldValue *tmp_Hx = new FieldValue [sizeHx];
  FieldValue *tmp_Hy = new FieldValue [sizeHy];

  FieldValue *tmp_Ez_prev = new FieldValue [sizeEz];
  FieldValue *tmp_Hx_prev = new FieldValue [sizeHx];
  FieldValue *tmp_Hy_prev = new FieldValue [sizeHy];

  FieldValue *tmp_eps = new FieldValue [sizeEps];
  FieldValue *tmp_mu = new FieldValue [sizeMu];

  for (grid_iter i = 0; i < sizeEz; ++i)
  {
    FieldPointValue* valEz = Ez.getFieldPointValue (i);
    tmp_Ez[i] = valEz->getCurValue ();
    tmp_Ez_prev[i] = valEz->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeHx; ++i)
  {
    FieldPointValue* valHx = Hx.getFieldPointValue (i);
    tmp_Hx[i] = valHx->getCurValue ();
    tmp_Hx_prev[i] = valHx->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeHy; ++i)
  {
    FieldPointValue* valHy = Hy.getFieldPointValue (i);
    tmp_Hy[i] = valHy->getCurValue ();
    tmp_Hy_prev[i] = valHy->getPrevValue ();
  }

  for (grid_iter i = 0; i < sizeEps; ++i)
  {
    FieldPointValue *valEps = Eps.getFieldPointValue (i);
    tmp_eps[i] = valEps->getCurValue ();
  }

  for (grid_iter i = 0; i < sizeMu; ++i)
  {
    FieldPointValue *valMu = Mu.getFieldPointValue (i);
    tmp_mu[i] = valMu->getCurValue ();
  }

  FieldValue *Ez_cuda;
  FieldValue *Hx_cuda;
  FieldValue *Hy_cuda;

  FieldValue *Ez_cuda_prev;
  FieldValue *Hx_cuda_prev;
  FieldValue *Hy_cuda_prev;

  FieldValue *eps_cuda;
  FieldValue *mu_cuda;

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda, sizeEzRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda, sizeHxRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda, sizeHyRaw));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda_prev, sizeEzRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda_prev, sizeHxRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda_prev, sizeHyRaw));

  cudaCheckErrorCmd (hipMalloc ((void **) &eps_cuda, sizeEpsRaw));
  cudaCheckErrorCmd (hipMalloc ((void **) &mu_cuda, sizeMuRaw));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda, tmp_Ez, sizeEzRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda, tmp_Hx, sizeHxRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda, tmp_Hy, sizeHyRaw, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda_prev, tmp_Ez_prev, sizeEzRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda_prev, tmp_Hx_prev, sizeHxRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda_prev, tmp_Hy_prev, sizeHyRaw, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (eps_cuda, tmp_eps, sizeEpsRaw, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (mu_cuda, tmp_mu, sizeMuRaw, hipMemcpyHostToDevice));

  CudaExitStatus exitStatus;
  CudaExitStatus *exitStatusCuda;
  cudaCheckErrorCmd (hipMalloc ((void **) &exitStatusCuda, sizeof (CudaExitStatus)));

  GridCoordinate3D EzStart = yeeLayout.getEzStart (Ez.getStart ());
  GridCoordinate3D EzEnd = yeeLayout.getEzEnd (Ez.getEnd ());

  GridCoordinate3D HxStart = yeeLayout.getHxStart (Hx.getStart ());
  GridCoordinate3D HxEnd = yeeLayout.getHxEnd (Hx.getEnd ());

  GridCoordinate3D HyStart = yeeLayout.getHyStart (Hy.getStart ());
  GridCoordinate3D HyEnd = yeeLayout.getHyEnd (Hy.getEnd ());

  dim3 blocksEz (EzSizeCoord.getX () / 16, EzSizeCoord.getY () / 16);
  dim3 threadsEz (16, 16);

  dim3 blocksHx (HxSizeCoord.getX () / 16, HxSizeCoord.getY () / 16);
  dim3 threadsHx (16, 16);

  dim3 blocksHy (HySizeCoord.getX () / 16, HySizeCoord.getY () / 16);
  dim3 threadsHy (16, 16);

  for (time_step t = stepStart; t < stepEnd; ++t)
  {
    cudaCheckExitStatus (cudaCalculateTMzEzStep <<< blocksEz, threadsEz >>> (exitStatusCuda,
                                                                             Ez_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hx_cuda_prev,
                                                                             Hy_cuda_prev,
                                                                             eps_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             EzStart.getX (),
                                                                             EzStart.getY (),
                                                                             EzEnd.getX (),
                                                                             EzEnd.getY (),
                                                                             EzSizeCoord.getX (),
                                                                             EzSizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzEzSource <<< blocksEz, threadsEz >>> (exitStatusCuda,
                                                                               Ez_cuda_prev,
                                                                               EzStart.getX (),
                                                                               EzStart.getY (),
                                                                               EzEnd.getX (),
                                                                               EzEnd.getY (),
                                                                               EzSizeCoord.getX (),
                                                                               EzSizeCoord.getY (),
                                                                               t));

    cudaCheckExitStatus (cudaCalculateTMzHxStep <<< blocksHx, threadsHx >>> (exitStatusCuda,
                                                                             Hx_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hx_cuda_prev,
                                                                             mu_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             HxStart.getX (),
                                                                             HxStart.getY (),
                                                                             HxEnd.getX (),
                                                                             HxEnd.getY (),
                                                                             HxSizeCoord.getX (),
                                                                             HxSizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzHyStep <<< blocksHy, threadsHy >>> (exitStatusCuda,
                                                                             Hy_cuda,
                                                                             Ez_cuda_prev,
                                                                             Hy_cuda_prev,
                                                                             mu_cuda,
                                                                             gridTimeStep,
                                                                             gridStep,
                                                                             HyStart.getX (),
                                                                             HyStart.getY (),
                                                                             HyEnd.getX (),
                                                                             HyEnd.getY (),
                                                                             HySizeCoord.getX (),
                                                                             HySizeCoord.getY (),
                                                                             t));

    cudaCheckExitStatus (cudaCalculateTMzHxSource <<< blocksHx, threadsHx >>> (exitStatusCuda,
                                                                               Hx_cuda_prev,
                                                                               HxStart.getX (),
                                                                               HxStart.getY (),
                                                                               HxEnd.getX (),
                                                                               HxEnd.getY (),
                                                                               HxSizeCoord.getX (),
                                                                               HxSizeCoord.getY (),
                                                                               t));

    cudaCheckExitStatus (cudaCalculateTMzHySource <<< blocksHy, threadsHy >>> (exitStatusCuda,
                                                                               Hy_cuda_prev,
                                                                               HyStart.getX (),
                                                                               HyStart.getY (),
                                                                               HyEnd.getX (),
                                                                               HyEnd.getY (),
                                                                               HySizeCoord.getX (),
                                                                               HySizeCoord.getY (),
                                                                               t));
  }

  cudaCheckErrorCmd (hipMemcpy (tmp_Ez, Ez_cuda, sizeEzRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hx, Hx_cuda, sizeHxRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hy, Hy_cuda, sizeHyRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (tmp_Ez_prev, Ez_cuda_prev, sizeEzRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hx_prev, Hx_cuda_prev, sizeHxRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_Hy_prev, Hy_cuda_prev, sizeHyRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (tmp_eps, eps_cuda, sizeEpsRaw, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (tmp_mu, mu_cuda, sizeMuRaw, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipFree (Ez_cuda));
  cudaCheckErrorCmd (hipFree (Hx_cuda));
  cudaCheckErrorCmd (hipFree (Hy_cuda));

  cudaCheckErrorCmd (hipFree (Ez_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hx_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hy_cuda_prev));

  cudaCheckErrorCmd (hipFree (eps_cuda));
  cudaCheckErrorCmd (hipFree (mu_cuda));

  for (grid_iter i = 0; i < sizeEz; ++i)
  {
    FieldPointValue* valEz = Ez.getFieldPointValue (i);
    valEz->setCurValue (tmp_Ez[i]);
    valEz->setPrevValue (tmp_Ez_prev[i]);
  }

  for (grid_iter i = 0; i < sizeHx; ++i)
  {
    FieldPointValue* valHx = Hx.getFieldPointValue (i);
    valHx->setCurValue (tmp_Hx[i]);
    valHx->setPrevValue (tmp_Hx_prev[i]);
  }

  for (grid_iter i = 0; i < sizeHy; ++i)
  {
    FieldPointValue* valHy = Hy.getFieldPointValue (i);
    valHy->setCurValue (tmp_Hy[i]);
    valHy->setPrevValue (tmp_Hy_prev[i]);
  }

  delete[] tmp_Ez;
  delete[] tmp_Hx;
  delete[] tmp_Hy;

  delete[] tmp_Ez_prev;
  delete[] tmp_Hx_prev;
  delete[] tmp_Hy_prev;

  delete[] tmp_eps;
  delete[] tmp_mu;

  *retval = CUDA_OK;
  return;
}

#endif
