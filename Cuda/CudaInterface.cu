#include "CudaInterface.h"
#include "CudaGlobalKernels.h"

void cudaExecuteTMzSteps (CudaExitStatus *retval,
                          FieldValue *Ez, FieldValue *Hx, FieldValue *Hy,
                          FieldValue *Ez_prev, FieldValue *Hx_prev, FieldValue *Hy_prev,
                          FieldValue gridTimeStep, FieldValue gridStep,
                          grid_coord sx, grid_coord sy,
                          time_step stepStart, time_step stepEnd,
                          uint32_t blocksX, uint32_t blocksY, uint32_t threadsX, uint32_t threadsY)
{
  FieldValue *Ez_cuda;
  FieldValue *Hx_cuda;
  FieldValue *Hy_cuda;

  FieldValue *Ez_cuda_prev;
  FieldValue *Hx_cuda_prev;
  FieldValue *Hy_cuda_prev;

  grid_iter size = (grid_iter) sx * sy * sizeof (FieldValue);
  //printf ("%llu=%ld*%ld*%lld", size, sx, sy, sizeof (FieldValue));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda, size));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda_prev, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda_prev, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda_prev, size));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda, Ez, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda, Hx, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda, Hy, size, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda_prev, Ez_prev, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda_prev, Hx_prev, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda_prev, Hy_prev, size, hipMemcpyHostToDevice));

  dim3 blocks (blocksX, blocksY);
  dim3 threads (threadsX, threadsY);

  CudaExitStatus exitStatus;
  CudaExitStatus *exitStatusCuda;
  cudaCheckErrorCmd (hipMalloc ((void **) &exitStatusCuda, sizeof (CudaExitStatus)));

  for (time_step t = stepStart; t < stepEnd; ++t)
  {
    cudaCheckExitStatus (cudaCalculateTMzEStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Ez_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx,
                                                                        sy,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzESource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Ez_cuda_prev,
                                                                          sx,
                                                                          sy,
                                                                          t));

    cudaCheckExitStatus (cudaCalculateTMzHStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Hx_cuda,
                                                                        Hy_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx,
                                                                        sy,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzHSource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Hx_cuda_prev,
                                                                          Hy_cuda_prev,
                                                                          sx,
                                                                          sy,
                                                                          t));
  }

  cudaCheckErrorCmd (hipMemcpy (Ez, Ez_cuda, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx, Hx_cuda, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy, Hy_cuda, size, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (Ez_prev, Ez_cuda_prev, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx_prev, Hx_cuda_prev, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy_prev, Hy_cuda_prev, size, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipFree (Ez_cuda));
  cudaCheckErrorCmd (hipFree (Hx_cuda));
  cudaCheckErrorCmd (hipFree (Hy_cuda));

  cudaCheckErrorCmd (hipFree (Ez_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hx_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hy_cuda_prev));

  *retval = CUDA_OK;
  return;
}
