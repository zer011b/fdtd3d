#include "CudaInterface.h"

void cudaExecute2DTMzSteps (CudaExitStatus *retval,
                            FieldValue *Ez, FieldValue *Hx, FieldValue *Hy,
                            FieldValue *Ez_prev, FieldValue *Hx_prev, FieldValue *Hy_prev,
                            FieldValue *eps, FieldValue *mu,
                            FieldValue gridTimeStep, FieldValue gridStep,
                            grid_coord sx, grid_coord sy,
                            time_step stepStart, time_step stepEnd,
                            uint32_t blocksX, uint32_t blocksY, uint32_t threadsX, uint32_t threadsY)
{
  FieldValue *Ez_cuda;
  FieldValue *Hx_cuda;
  FieldValue *Hy_cuda;

  FieldValue *Ez_cuda_prev;
  FieldValue *Hx_cuda_prev;
  FieldValue *Hy_cuda_prev;

  FieldValue *eps_cuda;
  FieldValue *mu_cuda;

  grid_iter size = (grid_iter) sx * sy * sizeof (FieldValue);
  //printf ("%llu=%ld*%ld*%lld", size, sx, sy, sizeof (FieldValue));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda, size));

  cudaCheckErrorCmd (hipMalloc ((void **) &Ez_cuda_prev, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hx_cuda_prev, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &Hy_cuda_prev, size));

  cudaCheckErrorCmd (hipMalloc ((void **) &eps_cuda, size));
  cudaCheckErrorCmd (hipMalloc ((void **) &mu_cuda, size));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda, Ez, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda, Hx, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda, Hy, size, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (Ez_cuda_prev, Ez_prev, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hx_cuda_prev, Hx_prev, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (Hy_cuda_prev, Hy_prev, size, hipMemcpyHostToDevice));

  cudaCheckErrorCmd (hipMemcpy (eps_cuda, eps, size, hipMemcpyHostToDevice));
  cudaCheckErrorCmd (hipMemcpy (mu_cuda, mu, size, hipMemcpyHostToDevice));

  dim3 blocks (blocksX, blocksY);
  dim3 threads (threadsX, threadsY);

  CudaExitStatus exitStatus;
  CudaExitStatus *exitStatusCuda;
  cudaCheckErrorCmd (hipMalloc ((void **) &exitStatusCuda, sizeof (CudaExitStatus)));

  for (time_step t = stepStart; t < stepEnd; ++t)
  {
    cudaCheckExitStatus (cudaCalculateTMzEStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Ez_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        eps_cuda,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx,
                                                                        sy,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzESource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Ez_cuda_prev,
                                                                          sx,
                                                                          sy,
                                                                          t));

    cudaCheckExitStatus (cudaCalculateTMzHStep <<< blocks, threads >>> (exitStatusCuda,
                                                                        Hx_cuda,
                                                                        Hy_cuda,
                                                                        Ez_cuda_prev,
                                                                        Hx_cuda_prev,
                                                                        Hy_cuda_prev,
                                                                        mu_cuda,
                                                                        gridTimeStep,
                                                                        gridStep,
                                                                        sx,
                                                                        sy,
                                                                        t));

    cudaCheckExitStatus (cudaCalculateTMzHSource <<< blocks, threads >>> (exitStatusCuda,
                                                                          Hx_cuda_prev,
                                                                          Hy_cuda_prev,
                                                                          sx,
                                                                          sy,
                                                                          t));
  }

  cudaCheckErrorCmd (hipMemcpy (Ez, Ez_cuda, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx, Hx_cuda, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy, Hy_cuda, size, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (Ez_prev, Ez_cuda_prev, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hx_prev, Hx_cuda_prev, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (Hy_prev, Hy_cuda_prev, size, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipMemcpy (eps, eps_cuda, size, hipMemcpyDeviceToHost));
  cudaCheckErrorCmd (hipMemcpy (mu, mu_cuda, size, hipMemcpyDeviceToHost));

  cudaCheckErrorCmd (hipFree (Ez_cuda));
  cudaCheckErrorCmd (hipFree (Hx_cuda));
  cudaCheckErrorCmd (hipFree (Hy_cuda));

  cudaCheckErrorCmd (hipFree (Ez_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hx_cuda_prev));
  cudaCheckErrorCmd (hipFree (Hy_cuda_prev));

  cudaCheckErrorCmd (hipFree (eps_cuda));
  cudaCheckErrorCmd (hipFree (mu_cuda));

  *retval = CUDA_OK;
  return;
}
