#include "hip/hip_runtime.h"
#include "invoke.h"

#include "cstdio"

__global__ void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        c[index] = a[index] + b[index];
}

void execute ()
{
  int N = 10;
  int *a, *b, *c;
int *d_a, *d_b, *d_c;
int size = N * sizeof( int );

int THREADS_PER_BLOCK = 1;

/* allocate space for device copies of a, b, c */

hipMalloc( (void **) &d_a, size );
hipMalloc( (void **) &d_b, size );
hipMalloc( (void **) &d_c, size );

/* allocate space for host copies of a, b, c and setup input values */

a = (int *)malloc( size );
b = (int *)malloc( size );
c = (int *)malloc( size );

for( int i = 0; i < N; i++ )
{
  a[i] = b[i] = i;
  c[i] = 0;
}

/* copy inputs to device */
/* fix the parameters needed to copy data to the device */
hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

/* launch the kernel on the GPU */
/* insert the launch parameters to launch the kernel properly using blocks and threads */
vector_add<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

/* copy result back to host */
/* fix the parameters needed to copy data back to the host */
hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


printf( "c[%d] = %d\n",0,c[0] );
printf( "c[%d] = %d\n",N-1, c[N-1] );

  printf( "a[%d] = %d\n",0,a[0] );
printf( "a[%d] = %d\n",N-1, a[N-1] );

/* clean up */

free(a);
free(b);
free(c);
hipFree( d_a );
hipFree( d_b );
hipFree( d_c );
}
